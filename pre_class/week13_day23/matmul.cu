
#include <hip/hip_runtime.h>
#include<iostream>
using std::cout; using std::endl;
#include<random>
using std::mt19937_64; using std::uniform_real_distribution; using std::random_device;
#include<algorithm>
using std::generate; using std::fill_n;
#include<chrono>
using namespace std::chrono_literals;

void fill_ary(float *ary, long cnt){
  mt19937_64 reng(random_device{}() );
  uniform_real_distribution<float> dist(0,10);
  generate(ary, ary+cnt, [&](){return dist(reng);});
}

void matmul_cpu (float *A, float *B, float *C, int size){
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      C[i*size + j] = 0;
      for (int k = 0; k < size; k++) {
	C[i*size +j] += A[i*size + k] * B[k*size + j];
      }
    }
  }
}

__global__ void matmul_basic(float* A, float* B, float* C, int width) {
  //calculate the row and column for this element of the matrix
  int row = threadIdx.y + (blockDim.y * blockIdx.y);
  int col = threadIdx.x + (blockDim.x * blockIdx.x);
  
  if ((row < width) && (col < width)) {
    float result = 0;
    for (int k = 0; k < width; k++) {
      result += A[(row * width) + k] * B[(k * width) + col];
    }
    C[(row * width) + col] = result;
  }
}

const int TILE_WIDTH = 32;
__global__ void matmul_shared(float* A, float* B, float* C, int width) {
  __shared__ float Ashare[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bshare[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  //calculate the row and column for this element of the matrix
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  
  float result = 0;
  //loop over the A and B tiles required to compute the C element
  for (int m = 0; m < width / TILE_WIDTH; m++) {
    /*
      Indexing parameters:
      For A:
                       row * width : Index global row for this thread (invariant)
	            m * TILE_WIDTH : Index new columns for each iter
	                        tx : Index column of this set
      For B:
            m * TILE_WIDTH * width : index, new set of rows each iter
	                ty * width : index row in that set
			       col : the global column (invariant)
    */				
    Ashare[ty][tx] = A[(row * width) + (m * TILE_WIDTH + tx)];
    Bshare[ty][tx] = B[(m * TILE_WIDTH * width) + (ty * width) + col];
    __syncthreads(); //wait for all the shared memory to be loaded

    for (int k = 0; k < TILE_WIDTH; k++) {
      result +=	Ashare[ty][k] * Bshare[k][tx];
    }
    __syncthreads(); //make sure all threads have done their calculation
                     //before modifying the shared memory.
  }
  C[(row * width) + col] = result;
}


int main (int argc, char **argv){
  if (argc != 2){
    cout << "Need a size"<<endl;
    exit(1);
  }
  int n = atoi(argv[1]);
  long sz = n*n;
  float *A_host, *B_host, *C_host, *A_dev, *B_dev, *C_dev;
  int nerror;

  int block_cnt, thread_cnt;

  A_host = new float[sz];
  B_host = new float[sz];
  C_host = new float[sz];

  std::chrono::time_point<std::chrono::steady_clock> start, stop;
  using time_span = std::chrono::milliseconds;

  // fill_ary(x_host, n);
  // fill_ary(y_host, n);

  // just to check our results.
  fill_n(A_host, sz, 1.0);
  fill_n(B_host, sz, 2.0);
  
  int bytes = sz * sizeof(float);
  hipMalloc((void **) &A_dev, bytes);
  hipMalloc((void **) &B_dev, bytes);
  hipMalloc((void **) &C_dev, bytes);

  hipMemcpy(A_dev, A_host, bytes, hipMemcpyHostToDevice);
  hipMemcpy(B_dev, B_host, bytes, hipMemcpyHostToDevice);

  thread_cnt = 512;
  block_cnt = sz/thread_cnt + (sz % thread_cnt > 0);

  start = std::chrono::steady_clock::now();
  for (int i=0; i<10; ++i)
    matmul_basic<<<block_cnt, thread_cnt>>>(A_dev, B_dev, C_dev, n);
  //  cudaDeviceSynchronize();
  hipMemcpy(C_host, C_dev, bytes, hipMemcpyDeviceToHost);
  stop = std::chrono::steady_clock::now();
  auto elapsed_gpu_1 = std::chrono::duration_cast<time_span>(stop - start).count();

  start = std::chrono::steady_clock::now();
  for (int i=0; i<10; ++i)
    matmul_shared<<<block_cnt, thread_cnt>>>(A_dev, B_dev, C_dev, n);
  // cudaDeviceSynchronize();
  hipMemcpy(C_host, C_dev, bytes, hipMemcpyDeviceToHost);
  stop = std::chrono::steady_clock::now();
  auto elapsed_gpu_2 = std::chrono::duration_cast<time_span>(stop - start).count();
  
  /* start = std::chrono::steady_clock::now(); */
  /* for (int i=0; i<10; ++i) */
  /*   matmul_cpu(A_host, B_host, C_host, n); */
  /* stop = std::chrono::steady_clock::now(); */
  /* auto elapsed_cpu = std::chrono::duration_cast<time_span>(stop - start).count(); */
  cout <<", GPU basic: "<<elapsed_gpu_1 <<", GPU shared: "<<elapsed_gpu_2<<endl;
  // cout << "CPU: "<<elapsed_cpu<<", GPU basic: "<<elapsed_gpu_1 <<", GPU shared: "<<elapsed_gpu_2<<endl;
}
