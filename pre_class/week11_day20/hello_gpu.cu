
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ functions, or "kernels", execute on the device
__global__ void hello_kernel(void)
{
  printf("Hello from GPU, I'm thread %d on block %d!\n", threadIdx.x, blockIdx.x);
  
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  hello_kernel<<<2,2>>>();

  // wait for the device to finish so that we see the message
  hipError_t cuda_err = hipDeviceSynchronize();
  if (cuda_err != hipSuccess)
    printf("launch error \"%s\".\n", hipGetErrorString(cuda_err));

  return 0;
}
