#include "hip/hip_runtime.h"

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


__global__ void scalarMulAdd(const float *A, const float *B, const float *C, float *D, int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) 
  {
    D[i] = A[i]*B[0] + C[0] + 0.0f;
  }
}

int main(void)
{
    hipError_t error = hipSuccess;

    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate CPU device
    // Allocate the host input vector A, D
    float *h_A = (float *)malloc(size);
    float *h_D = (float *)malloc(size);

    // Allocate the host input B and C
    float *h_B = (float *)malloc(sizeof(float));
    float *h_C = (float *)malloc(sizeof(float));

    //Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL || h_D == NULL ) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Allocate GPU device
    float *d_A = NULL;
    err_A = hipMalloc((void **)&d_A, size);
    float *d_D = NULL;
    err_D = hipMalloc((void **)&d_D, size);

    float *d_B = NULL;
    err_B = hipMalloc((void **)&d_B, sizeof(float));
    float *d_C = NULL;
    err_C = hipMalloc((void **)&d_C, sizeof(float));

    if (err_A != hipSuccess || err_B != hipSuccess || err_C != hipSuccess || err_D != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);

    }

    // Fill memory
    for (int i=0; i < numElements; ++i)
        h_A[i] = 1.0;
    h_B[0] = 2.0;
    h_C[0] = 3.0; 


    // Copy from host to device
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_B, h_B, sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(d_C, h_C, sizeof(float), hipMemcpyHostToDevice);


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
    scalarMulAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch scalarMulAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

    for (int i=0; i< numElements; ++i)
        printf("%d %d", h_A[i] * h_B[0] + h_C[0], h_D[0]);

    // Free device global memory
    err = hipFree(d_A);
    err = hipFree(d_B);
    err = hipFree(d_C);
    err = hipFree(d_D);
}
