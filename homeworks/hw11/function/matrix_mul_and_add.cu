
#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void matrixMulAdd(const float *A, const float *B, const float *C, float *D, int numElements)
{
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  int row = blockDim.y * blockIdx.y + threadIdx.y;

  float sum = 0.0f;

  // Do matrix multiplication first
  for (int k = 0; k < numElements; ++k)
    sum += A[row * numElements + k] * B[k];

  // Now add the third vector
  D[col] = sum +  C[col];
}

int main(int argc, char** argv)
{
    if (argc < 1){
        printf("a.out block_size");
        exit;
    }
    int blocksPerGrid = atoi(argv[1]);

    int numElements = 512;
    size_t size_bytes = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate CPU device
    // Allocate the host input vector A, B, C, D
    float *h_A = (float *)malloc(numElements * size_bytes);    
    float *h_B = (float *)malloc(size_bytes);
    float *h_C = (float *)malloc(size_bytes);
    float *h_D = (float *)malloc(size_bytes);

    //Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL || h_D == NULL ) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Allocate GPU device
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    float *d_D = NULL;

    hipMalloc((void **)&d_A, numElements * size_bytes);
    hipMalloc((void **)&d_B, size_bytes);
    hipMalloc((void **)&d_C, size_bytes);
    hipMalloc((void **)&d_D, size_bytes);

    // Fill memory
    for (int i=0; i < numElements; ++i)
    {
        for (int j=0; j < numElements; ++j)
            h_A[i*numElements + j] = 1.0;
        h_B[i] = 1.0;
        h_C[i] = 3.0; 
    }

    // Copy from host to device
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, numElements * size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_bytes, hipMemcpyHostToDevice);

    // Launch the matrixMulAdd CUDA Kernel
    int threadsPerBlock = (numElements + blocksPerGrid - 1) / blocksPerGrid;
    //int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
    matrixMulAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, numElements);

    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_D, d_D, size_bytes, hipMemcpyDeviceToHost);

    // Debug Printing
    // for (int i=0; i< numElements; ++i)
    //    printf("%f ", h_D[i]);

    // Free device global memory
    delete [] h_A;
    delete [] h_B;
    delete [] h_C;
    delete [] h_D;    

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return 0;
}
